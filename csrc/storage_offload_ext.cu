#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <pybind11/pybind11.h>
#include <c10/cuda/CUDAStream.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>

#include <iostream>
#include <algorithm>
#include <cassert>
#include <fstream>
#include <string>
#include <vector>
#include <chrono>
#include <map>
#include <future>
#include <mutex>
#include <thread>
#include <queue>
#include <condition_variable>
#include <memory>
#include <atomic>
#include <optional>
#include <sys/syscall.h>
#include <sys/stat.h>
#include <unistd.h>
#include <fcntl.h>
#include <cstring>
#include <cstdio>
#include <liburing.h>

namespace py = pybind11;

#define TIME_EXPR(label, expr) ([&]() { \
    auto __t0 = std::chrono::high_resolution_clock::now(); \
    auto __ret = (expr); \
    auto __t1 = std::chrono::high_resolution_clock::now(); \
    double __ms = std::chrono::duration<double, std::milli>(__t1 - __t0).count(); \
    std::cout << "[TIME] " << label << " took " << __ms << " ms\n"; \
    return __ret; \
})()
// -------------------------------------
// Thread-local pinned buffer
// -------------------------------------
static thread_local void* t_pinned_ptr = nullptr;
static thread_local size_t t_pinned_size = 0;

static std::pair<void*, size_t> get_thread_local_pinned(size_t required_bytes) {
    if (!t_pinned_ptr || t_pinned_size < required_bytes) {
        if (t_pinned_ptr) {
            hipHostFree(t_pinned_ptr);
            t_pinned_ptr = nullptr;
            t_pinned_size = 0;
        }

        size_t alloc_size = std::max(required_bytes, (size_t)16 * 1024 * 1024); // at least 16 MB
        hipError_t err = hipHostMalloc(&t_pinned_ptr, alloc_size);
        if (err != hipSuccess) {
            std::cerr << "[ERROR] hipHostMalloc failed: "
                      << hipGetErrorString(err) << "\n";
            t_pinned_ptr = nullptr;
            t_pinned_size = 0;
        } else {
            t_pinned_size = alloc_size;
            std::cout << "[INFO] Thread " << std::this_thread::get_id()
                      << " allocated pinned buffer "
                      << (alloc_size / (1024 * 1024)) << " MB\n";
        }
    }
    return {t_pinned_ptr, t_pinned_size};
}
// Tracks async job progress and results
struct JobState {
    // Futures for each async task in the job
    std::vector<std::shared_future<bool>> futures;
    // Number of tasks completed so far
    std::atomic<int> completed_tasks{0};
    // Total number of tasks scheduled for this job
    std::atomic<int> total_tasks{0};
    // Flag indicating if all tasks succeeded
    std::atomic<bool> all_success{true};
};
// --------------------------------------
// Global resources - TODO should removed all Global variables
// --------------------------------------
// CUDA streams assigned to each worker thread
static std::vector<at::cuda::CUDAStream> g_streams;

// Thread-local CUDA stream used by the current worker
static thread_local std::optional<c10::cuda::CUDAStream> thread_stream;
// Thread-local stream index for mapping worker threads to streams
static thread_local size_t thread_stream_idx = 0;

thread_local io_uring* t_ring = nullptr;

// Mutex protecting access to the jobs map
static std::mutex jobs_mutex;
// Global map of job_id → JobState, tracking async job progress
static std::map<int, std::unique_ptr<JobState>> jobs;

struct PinnedBufferInfo {
    void* ptr = nullptr;
    size_t size = 0;
};

static std::vector<PinnedBufferInfo> g_pinned_buffers;
void preallocate_pinned_buffers(size_t io_threads, size_t pinned_buffer_size_mb) {
    g_pinned_buffers.resize(io_threads);
    size_t alloc_bytes = pinned_buffer_size_mb * 1024 * 1024;

    for (size_t i = 0; i < io_threads; ++i) {
        void* ptr = nullptr;
        hipError_t err = hipHostMalloc(&ptr, alloc_bytes);
        if (err != hipSuccess) {
            std::cerr << "[ERROR] Failed to allocate pinned buffer for thread "
                      << i << ": " << hipGetErrorString(err) << std::endl;
            g_pinned_buffers[i] = {nullptr, 0};
        } else {
            g_pinned_buffers[i] = {ptr, alloc_bytes};

        }
    }

    std::cout << "[INFO] Pre-allocated pinned buffer "
            << (alloc_bytes / (1024 * 1024))
            << " MB for " << io_threads << " threads" << std::endl;
}

// -------------------------------
// I/O thread pool for scheduling
// -------------------------------
class IOThreadPool {
private:
    std::vector<std::thread> workers;
    std::queue<std::function<void()>> tasks;
    std::mutex queue_mutex;
    std::condition_variable condition;
    std::atomic<bool> stop{false};

public:
    IOThreadPool(int threads, size_t pinned_buffer_mb, int tp_rank) {
        // Initialize PyTorch threading globally (main thread only)
        at::init_num_threads();
        at::set_num_threads(1);

        for (size_t i = 0; i < threads; ++i) {
            workers.emplace_back([this, i, threads, pinned_buffer_mb, tp_rank] {
                // Compute unique CPU per thread
                int cpu_id = tp_rank * threads + i;
                cpu_set_t cpuset;
                CPU_ZERO(&cpuset);
                CPU_SET(cpu_id, &cpuset);

                if (pthread_setaffinity_np(pthread_self(), sizeof(cpuset), &cpuset) != 0) {
                    std::cerr << "[WARN] Failed to set affinity for thread " << i
                              << " tp_rank=" << tp_rank << " to CPU " << cpu_id << "\n";
                }

                pid_t tid = static_cast<pid_t>(syscall(SYS_gettid));
                int actual_cpu = sched_getcpu();

                std::cout << "[INFO] IO thread " << i
                          << " (tid=" << tid << ", tp_rank=" << tp_rank
                          << ") pinned to CPU " << cpu_id
                          << " (running on CPU " << actual_cpu << ")\n";

                if (i < g_pinned_buffers.size() && g_pinned_buffers[i].ptr != nullptr) {
                    t_pinned_ptr = g_pinned_buffers[i].ptr;
                    t_pinned_size = g_pinned_buffers[i].size;
                    std::cout << "[INFO] IO thread " << i
                            << " attached to preallocated pinned buffer "
                            << (t_pinned_size / (1024 * 1024)) << " MB\n";
                } else {
                    std::cerr << "[WARN] IO thread " << i
                            << " has no preallocated pinned buffer\n";
                }
                // Initialize io_uring for this thread
                if (!t_ring) {
                    t_ring = new io_uring();
                    int ret = io_uring_queue_init(256, t_ring, 0);
                    if (ret < 0) {
                        std::cerr << "[ERROR] Failed to initialize io_uring for thread "
                                << i << ": " << strerror(-ret) << std::endl;
                        t_ring = nullptr;
                    } else {
                        std::cout << "[INFO] io_uring initialized for thread "
                                << i << " with depth 256\n";
                    }
                }

                thread_stream_idx = i;
                while (true) {
                    std::function<void()> task;
                    {
                        std::unique_lock<std::mutex> lock(queue_mutex);
                        condition.wait(lock, [this] { return stop || !tasks.empty(); });
                        if (stop && tasks.empty()) return;
                        task = std::move(tasks.front());
                        tasks.pop();
                    }
                    task();
                }
            });
        }
    }

    ~IOThreadPool() {
        stop = true;
        condition.notify_all();
        for (std::thread &worker : workers) {
            worker.join();
        }
    }

    template<class F>
    auto enqueue(F&& f) -> std::future<typename std::result_of<F()>::type> {
        using return_type = typename std::result_of<F()>::type;

        auto task = std::make_shared<std::packaged_task<return_type()>>(
            std::forward<F>(f)
        );

        std::future<return_type> res = task->get_future();
        {
            std::unique_lock<std::mutex> lock(queue_mutex);
            if (stop) throw std::runtime_error("enqueue on stopped ThreadPool");
            tasks.emplace([task](){ (*task)(); });
        }
        condition.notify_one();
        return res;
    }
};

// Global IO thread pool for scheduling async PUT/GET tasks
static std::unique_ptr<IOThreadPool> g_io_pool;

// -------------------------------
// Initialize resources with pre-allocation
// -------------------------------

// Initialize IO threads, CUDA streams, and pinned memory pool
void init_performance_resources(int io_threads = 0, size_t pinned_buffer_size_mb = 256 ,size_t max_pinned_memory_gb = 10, int tp_rank = 1) {
    if (!g_io_pool) {
        if (io_threads == 0) {
            io_threads = std::max(4u, std::thread::hardware_concurrency() / 2);
        }
        std::cout << "[INFO] Initializing IOThreadPool with "
                  << io_threads << " threads, "
                  << pinned_buffer_size_mb << " MB pinned buffer per thread, "
                  << max_pinned_memory_gb << " GB max pinned memory\n"
                  << std::thread::hardware_concurrency() << " hardware_concurrency\n";

        // Pre-allocate pinned buffers before launching threads
        preallocate_pinned_buffers(io_threads, pinned_buffer_size_mb);

        g_io_pool = std::make_unique<IOThreadPool>(io_threads, pinned_buffer_size_mb, tp_rank);
        // Create dedicated streams for each thread
        g_streams.clear();
        g_streams.reserve(io_threads);
        for (size_t i = 0; i < io_threads; i++) {
            g_streams.push_back(at::cuda::getStreamFromPool(/*isHighPriority=*/false));
        }

        // Warm up CUDA context and streams
        for (auto& stream : g_streams) {
            hipStreamSynchronize(stream.stream());
        }
    }
}


// -------------------------------
// Status and cleanup
// -------------------------------
// Return finished jobs and their success status
std::vector<std::pair<int, bool>> get_finished_ext() {
    std::lock_guard<std::mutex> lock(jobs_mutex);

    std::vector<std::pair<int, bool>> results;
    std::vector<int> to_erase;

    // Iterate over all active jobs.
    for (auto &kv : jobs) {
        int job_id = kv.first;
        auto &job_state = kv.second;

        // Check if the job has completed all its tasks.
        if (job_state->completed_tasks.load() == job_state->total_tasks.load()) {
            bool all_ok = job_state->all_success.load();
            results.emplace_back(job_id, all_ok);
            to_erase.push_back(job_id);
        }
    }

    // Remove all finished jobs from the map.
    for (int jid : to_erase) {
        jobs.erase(jid);
    }
    return results;
}

// Release IO threads, CUDA streams, and pinned buffer
void cleanup_performance_resources() {
    g_io_pool.reset();
    g_streams.clear();

    if (t_pinned_ptr) {
        hipHostFree(t_pinned_ptr);
        t_pinned_ptr = nullptr;
        t_pinned_size = 0;
    }

    // Cleanup io_uring when thread exits
    if (t_ring) {
        io_uring_queue_exit(t_ring);
        delete t_ring;
        t_ring = nullptr;
    }
}

//----------------------------------------------------------------------
// GPU → Storage (PUT)
// ----------------------------------------------------------------------
// Copy selected GPU tensor blocks into pinned CPU buffer asynchronously
torch::Tensor copy_gpu_tensors_to_buffer(
    const std::vector<torch::Tensor>& src_tensors,
    const std::vector<int64_t>& block_ids_list,
    const c10::cuda::CUDAStream& stream) {

    auto dtype = src_tensors[0].dtype();
    size_t element_size = src_tensors[0].element_size();

    // Calculate dimensions
    size_t total_elements = 0;
    for (int64_t block_id : block_ids_list) {
        for (const auto &tensor : src_tensors) {
            auto block = tensor.index({torch::indexing::Slice(), block_id});
            total_elements += block.numel();
        }
    }

    size_t required_bytes = total_elements * element_size;
    auto [pinned_ptr, pinned_size] = get_thread_local_pinned(required_bytes);

    // Create output tensor view
    torch::Tensor result_cpu = torch::from_blob(
        pinned_ptr,
        {static_cast<long>(total_elements)},
        torch::TensorOptions().dtype(dtype).device(torch::kCPU).pinned_memory(true)
    );

    // Direct async copy without intermediate cat
    size_t offset = 0;
    for (int64_t block_id : block_ids_list) {
        for (const auto &tensor : src_tensors) {
            auto block = tensor.index({torch::indexing::Slice(), block_id}).contiguous();
            size_t block_bytes = block.nbytes();

            hipMemcpyAsync(
                static_cast<char*>(pinned_ptr) + offset,
                block.data_ptr(),
                block_bytes,
                hipMemcpyDeviceToHost,
                stream.stream()
            );

            offset += block_bytes;
        }
    }

    if (result_cpu.dtype() == torch::kBFloat16) {
        result_cpu = result_cpu.view(torch::kUInt16);
    }

    return result_cpu;
}

// File write helpers - optimized for large writes
bool write_file_to_disk(const std::string &target_path,
                                 const torch::Tensor &host_buf,
                                 io_uring* ring) {
    const void* data_ptr = host_buf.data_ptr();
    size_t nbytes = host_buf.nbytes();
    std::string tmp_path = target_path + ".tmp";

    int fd = ::open(tmp_path.c_str(), O_WRONLY | O_CREAT | O_TRUNC, 0644);
    if (fd < 0) {
        std::cerr << "[ERROR] open failed: " << tmp_path << "\n";
        return false;
    }

    struct io_uring_sqe* sqe = io_uring_get_sqe(ring);
    if (!sqe) {
        std::cerr << "[ERROR] io_uring_get_sqe failed\n";
        ::close(fd);
        return false;
    }

    io_uring_prep_write(sqe, fd, data_ptr, nbytes, 0);
    sqe->flags |= IOSQE_IO_LINK; // optional: chain writes if batching

    int ret = io_uring_submit(ring);
    if (ret < 0) {
        std::cerr << "[ERROR] io_uring_submit failed: " << strerror(-ret) << "\n";
        ::close(fd);
        return false;
    }

    struct io_uring_cqe* cqe;
    ret = io_uring_wait_cqe(ring, &cqe);
    if (ret < 0 || cqe->res < 0) {
        std::cerr << "[ERROR] io_uring write failed: " << strerror(-cqe->res) << "\n";
        io_uring_cqe_seen(ring, cqe);
        ::close(fd);
        return false;
    }

    io_uring_cqe_seen(ring, cqe);
    ::close(fd);

    std::rename(tmp_path.c_str(), target_path.c_str());
    return true;
}

// Async GPU → Storage transfer (PUT)
bool transfer_async_put_ext(int job_id,
                        std::vector<std::string> target_files,
                        std::vector<torch::Tensor> src_tensors,
                        std::vector<std::vector<int64_t>> all_block_ids) {

    // Create job state object that will track progress and futures for this job.
    auto job_state = std::make_unique<JobState>();
    job_state->total_tasks = target_files.size();

    // For each target file, enqueue one async task in the I/O thread pool.
    for (size_t i = 0; i < target_files.size(); i++) {
        std::string target = target_files[i];
        auto src = src_tensors;
        auto bids = all_block_ids[i];

        auto future = g_io_pool->enqueue([=, job_state = job_state.get()]() -> bool {

          // Each thread gets a dedicated CUDA stream for async GPU ops.
            if (!thread_stream.has_value()) {
                thread_stream = g_streams[thread_stream_idx % g_streams.size()];
            }

            // Save current CUDA stream so we can restore it later.
            auto current_stream = at::cuda::getCurrentCUDAStream();
            at::cuda::setCurrentCUDAStream(*thread_stream);

            try {
                // Stage 1: Asynchronously copy tensors from GPU to pinned CPU buffer.
                auto host_buf = copy_gpu_tensors_to_buffer(src, bids, *thread_stream);

                // Stage 2: Synchronize only this thread's CUDA stream (not all).
                hipStreamSynchronize(thread_stream->stream());

                // Stage 3: Write the pinned buffer to disk (blocking operation).
                //bool ok = TIME_EXPR("write_file_to_disk", write_file_to_disk(target, host_buf));
                bool ok = write_file_to_disk(target, host_buf, t_ring);

                if (!ok)
                    std::cerr << "[ERROR] PUT failed during file write: " << target << "\n";

                // Restore original CUDA stream for safety.
                at::cuda::setCurrentCUDAStream(current_stream);

                // Atomically mark task completion.
                job_state->completed_tasks.fetch_add(1);
                if (!ok) job_state->all_success = false;
                return ok;

            } catch (...) {
                at::cuda::setCurrentCUDAStream(current_stream);
                job_state->completed_tasks.fetch_add(1);
                job_state->all_success = false;
                std::cerr << "[ERROR] PUT failed for " << target << "\n";
                return false;
            }
        });

        // Convert std::future → std::shared_future- is copyable and can be waited on by multiple threads.
        job_state->futures.push_back(future.share());
    }

    std::lock_guard<std::mutex> lock(jobs_mutex); // protect jobs map
    jobs[job_id] = std::move(job_state);

    return true;
}
// ----------------------------------------------------------------------
// Storage -> GPU (GET)
// ----------------------------------------------------------------------

// Read a file into a pinned CPU tensor from the pool
torch::Tensor read_file_to_pinned_tensor(const std::string& path, io_uring* ring) {
    if (ring == nullptr) {
        std::cerr << "[ERROR] io_uring ring is null for thread when reading " << path << "\n";
        throw std::runtime_error("io_uring ring not initialized");
    }

    // Get file size via stat
    struct stat st;
    if (::stat(path.c_str(), &st) != 0) {
        std::cerr << "[ERROR] stat failed for " << path << ": " << strerror(errno) << "\n";
        throw std::runtime_error("stat failed for " + path);
    }
    size_t file_size = static_cast<size_t>(st.st_size);

    // Allocate pinned buffer
    auto [pinned_ptr, pinned_size] = get_thread_local_pinned(file_size);
    if (!pinned_ptr || pinned_size < file_size) {
        std::cerr << "[ERROR] pinned buffer too small for file " << path
                  << " need " << file_size << " have " << pinned_size << "\n";
        throw std::runtime_error("pinned buffer too small");
    }

    // Open the file
    int fd = ::open(path.c_str(), O_RDONLY);
    if (fd < 0) {
        std::cerr << "[ERROR] open failed for " << path << ": " << strerror(errno) << "\n";
        throw std::runtime_error("open failed for " + path);
    }

    // Prepare and submit read
    io_uring_sqe* sqe = io_uring_get_sqe(ring);
    if (!sqe) {
        ::close(fd);
        std::cerr << "[ERROR] io_uring_get_sqe failed for " << path << "\n";
        throw std::runtime_error("io_uring_get_sqe failed");
    }

    io_uring_prep_read(sqe, fd, pinned_ptr, file_size, 0);

    int ret = io_uring_submit(ring);
    if (ret < 0) {
        ::close(fd);
        std::cerr << "[ERROR] io_uring_submit failed: " << strerror(-ret) << "\n";
        throw std::runtime_error("io_uring_submit failed");
    }

    // Wait for completion
    io_uring_cqe* cqe = nullptr;
    ret = io_uring_wait_cqe(ring, &cqe);
    if (ret < 0 || cqe->res < 0) {
        const int err = (ret < 0) ? -ret : -cqe->res;
        if (cqe) io_uring_cqe_seen(ring, cqe);
        ::close(fd);
        std::cerr << "[ERROR] io_uring read failed for " << path << ": " << strerror(err) << "\n";
        throw std::runtime_error("io_uring read failed");
    }

    io_uring_cqe_seen(ring, cqe);
    ::close(fd);

    // Wrap pinned buffer into a Torch tensor (no copy)
    auto options = torch::TensorOptions()
        .dtype(torch::kUInt8)
        .device(torch::kCPU)
        .pinned_memory(true);

    return torch::from_blob(
        pinned_ptr,
        {static_cast<long>(file_size)},
        [pinned_ptr](void* /*unused*/) {},  // no-op deleter
        options
    );
}

// Copy buffer to GPU tensors asynchronously using hipMemcpyAsync (no swap_blocks)
// CPU layout (contiguous): [num_blocks_in_file, num_layers, 2, H, B, D]
// GPU layout (contiguous): [2, num_blocks_total, H, B, D]
bool copy_buffer_to_gpu_tensors(
    torch::Tensor cpu_buf,
    const std::vector<int64_t>& block_ids_list,
    const std::vector<torch::Tensor>& dst_tensors,
    int num_blocks_in_file,
    const c10::cuda::CUDAStream& stream) {

    TORCH_CHECK(!dst_tensors.empty(), "Destination tensors list is empty");
    const auto& ref = dst_tensors[0];
    TORCH_CHECK(ref.is_contiguous(), "dst_tensors must be contiguous");
    TORCH_CHECK(cpu_buf.is_contiguous(), "cpu buffer must be contiguous");

    const auto shape = ref.sizes();  // [2, num_blocks_total, H, B, D]
    TORCH_CHECK(shape.size() == 5, "Expected shape [2, num_blocks, H, B, D]");

    const int64_t num_blocks_tot = shape[1];
    const int64_t H = shape[2];
    const int64_t B = shape[3];
    const int64_t D = shape[4];
    const int num_layers = static_cast<int>(dst_tensors.size());

    const size_t elem_size = ref.element_size();
    const size_t bytes_per_plane = static_cast<size_t>(H * B * D) * elem_size;
    const size_t bytes_per_block = 2 * bytes_per_plane;  // [K,V] in CPU buf

    auto* cpu_base = cpu_buf.data_ptr<uint8_t>();

    for (size_t bi = 0; bi < block_ids_list.size(); ++bi) {
        const int64_t gblock = block_ids_list[bi];
        const int64_t lblock = (num_blocks_in_file > 0) ? (gblock % num_blocks_in_file) : 0;
        const size_t cpu_block_base = static_cast<size_t>(lblock) * num_layers * bytes_per_block;

        for (int layer = 0; layer < num_layers; ++layer) {
            const auto& layer_tensor = dst_tensors[layer];
            auto* dst_base = reinterpret_cast<uint8_t*>(layer_tensor.data_ptr());

            // Compute CPU source offsets for K and V
            const size_t cpu_K_off = cpu_block_base + static_cast<size_t>(layer) * bytes_per_block;
            const size_t cpu_V_off = cpu_K_off + bytes_per_plane;

            // Compute GPU destination offsets for K and V
            const size_t gpu_K_off = static_cast<size_t>(gblock) * bytes_per_plane;
            const size_t gpu_V_off = (static_cast<size_t>(num_blocks_tot) + gblock) * bytes_per_plane;

            void* dst_K = dst_base + gpu_K_off;
            void* dst_V = dst_base + gpu_V_off;
            void* src_K = cpu_base + cpu_K_off;
            void* src_V = cpu_base + cpu_V_off;


            hipError_t err1 = hipMemcpyAsync(
                dst_K, src_K, bytes_per_plane,
                hipMemcpyHostToDevice, stream.stream());

            hipError_t err2 = hipMemcpyAsync(
                dst_V, src_V, bytes_per_plane,
                hipMemcpyHostToDevice, stream.stream());

            if (err1 != hipSuccess || err2 != hipSuccess) {
                std::cerr << "[ERROR] hipMemcpyAsync failed for block=" << gblock
                          << " layer=" << layer
                          << " err1=" << hipGetErrorString(err1)
                          << " err2=" << hipGetErrorString(err2)
                          << std::endl;
                return false;
            }
        }
    }

    return true;
}

// Async Storage → GPU transfer (GET)
bool transfer_async_get_ext(
    int job_id,
    std::vector<std::string> source_files,
    std::vector<std::vector<int64_t>> all_block_ids,
    std::vector<torch::Tensor> dst_tensors,
    int gpu_blocks_per_file)
{
    // Create job state object to track progress and futures for this job.
    auto job_state = std::make_unique<JobState>();
    job_state->total_tasks = source_files.size();

    // For each source file, enqueue one async task in the I/O thread pool.
    for (size_t i = 0; i < source_files.size(); i++) {
        std::string src_file = source_files[i];
        auto block_ids = all_block_ids[i];
        auto future = g_io_pool->enqueue([=, job_state = job_state.get()]() -> bool {

            // Get dedicated stream for this thread
            if (!thread_stream.has_value()) {
                thread_stream = g_streams[thread_stream_idx % g_streams.size()];
            }

            // Save current CUDA stream so we can restore it later.
            auto current_stream = at::cuda::getCurrentCUDAStream();
            at::cuda::setCurrentCUDAStream(*thread_stream);

            // -------------------------
            // Stage 1: File → pinned CPU
            // -------------------------
            bool stage1_ok = false;
            torch::Tensor host_buf;
            try {
                // Read data from disk into pinned memory tensor.
                //host_buf = read_file_to_pinned_tensor(src_file);
                host_buf = TIME_EXPR("read_file_to_pinned_tensor", read_file_to_pinned_tensor(src_file, t_ring));
                stage1_ok = true;
            } catch (const std::exception& e) {
                std::cerr << "[ERROR] Stage1 read_file_to_pinned_tensor failed for "
                        << src_file << ": " << e.what() << std::endl;
            } catch (...) {
                std::cerr << "[ERROR] Stage1 unknown failure for " << src_file << std::endl;
            }

            // -------------------------
            // Stage 2: CPU → GPU copy
            // -------------------------
            bool ok = false;
            if (stage1_ok) {
                try {
                    // Perform asynchronous GPU copy and tensor swap.
                    ok = copy_buffer_to_gpu_tensors(
                        host_buf, block_ids, dst_tensors, gpu_blocks_per_file, *thread_stream);

                    hipError_t err = hipStreamSynchronize(thread_stream->stream());
                    if (err != hipSuccess) {
                        std::cerr << "[ERROR] hipStreamSynchronize failed: "
                                << hipGetErrorString(err) << std::endl;
                        ok = false;
                    }

                } catch (const std::exception& e) {
                    std::cerr << "[ERROR] Stage2 copy_and_swap failed for " << src_file
                            << ": " << e.what() << std::endl;
                } catch (...) {
                    std::cerr << "[ERROR] Stage2 unknown failure for " << src_file << std::endl;
                }
            }

            // -------------------------
            // Final cleanup & accounting
            // -------------------------
            // Synchronize only this thread's CUDA stream.
            at::cuda::setCurrentCUDAStream(current_stream);
            job_state->completed_tasks.fetch_add(1);
            if (!ok) job_state->all_success = false;
            return ok;
        });
        // Convert std::future → std::shared_future- is copyable and can be waited on by multiple threads.

        job_state->futures.push_back(future.share());
    }

    std::lock_guard<std::mutex> lock(jobs_mutex);
    jobs[job_id] = std::move(job_state);
    return true;
}

// -------------------------------
// PYBIND11 module
// -------------------------------
PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("init_performance_resources", &init_performance_resources,
          py::arg("io_threads") = 0,
          py::arg("pinned_buffer_size_mb") = 256,
          py::arg("max_pinned_memory_gb") = 50,
          py::arg("tp_rank") = 1);

    m.def("cleanup_performance_resources", &cleanup_performance_resources);

    m.def("get_finished_ext", &get_finished_ext);

    m.def("transfer_async_put_ext", &transfer_async_put_ext,
          "Async transfer  GPU-> Storage",
          py::arg("job_id"),
          py::arg("target_files"),
          py::arg("all_src_tensors"),
          py::arg("all_block_ids"));

    m.def("transfer_async_get_ext", &transfer_async_get_ext,
      "Async GET transfer from Storage → GPU",
      py::arg("job_id"),
      py::arg("source_files"),
      py::arg("all_block_ids"),
      py::arg("dst_tensors"),
      py::arg("gpu_blocks_per_file"));
}
