#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <pybind11/pybind11.h>
#include <c10/cuda/CUDAStream.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>

#include <iostream>
#include <algorithm>
#include <cassert>
#include <fstream>
#include <string>
#include <vector>
#include <chrono>
#include <map>
#include <future>
#include <mutex>
#include <thread>
#include <queue>
#include <condition_variable>
#include <memory>
#include <atomic>
#include <optional>
#include <sys/syscall.h>
#include <sys/stat.h>
#include <unistd.h>
#include <fcntl.h>
#include <cstring>

namespace py = pybind11;

#define TIME_EXPR(label, expr) ([&]() { \
    auto __t0 = std::chrono::high_resolution_clock::now(); \
    auto __ret = (expr); \
    auto __t1 = std::chrono::high_resolution_clock::now(); \
    double __ms = std::chrono::duration<double, std::milli>(__t1 - __t0).count(); \
    std::cout << "[TIME] " << label << " took " << __ms << " ms\n"; \
    return __ret; \
})()
// -------------------------------------
// Thread-local pinned buffer
// -------------------------------------
static thread_local void* t_pinned_ptr = nullptr;
static thread_local size_t t_pinned_size = 0;

static std::pair<void*, size_t> get_thread_local_pinned(size_t required_bytes) {
    if (!t_pinned_ptr || t_pinned_size < required_bytes) {
        if (t_pinned_ptr) {
            hipHostFree(t_pinned_ptr);
            t_pinned_ptr = nullptr;
            t_pinned_size = 0;
        }

        size_t alloc_size = std::max(required_bytes, (size_t)16 * 1024 * 1024); // at least 16 MB
        hipError_t err = hipHostMalloc(&t_pinned_ptr, alloc_size);
        if (err != hipSuccess) {
            std::cerr << "[ERROR] hipHostMalloc failed: "
                      << hipGetErrorString(err) << "\n";
            t_pinned_ptr = nullptr;
            t_pinned_size = 0;
        } else {
            t_pinned_size = alloc_size;
            std::cout << "[INFO] Thread " << std::this_thread::get_id()
                      << " allocated pinned buffer "
                      << (alloc_size / (1024 * 1024)) << " MB\n";
        }
    }
    return {t_pinned_ptr, t_pinned_size};
}
// Tracks async job progress and results
struct JobState {
    // Futures for each async task in the job
    std::vector<std::shared_future<bool>> futures;
    // Number of tasks completed so far
    std::atomic<int> completed_tasks{0};
    // Total number of tasks scheduled for this job
    std::atomic<int> total_tasks{0};
    // Flag indicating if all tasks succeeded
    std::atomic<bool> all_success{true};
};
// --------------------------------------
// Global resources - TODO should removed all Global variables
// --------------------------------------
// CUDA streams assigned to each worker thread
static std::vector<at::cuda::CUDAStream> g_streams;

// Thread-local CUDA stream used by the current worker
static thread_local std::optional<c10::cuda::CUDAStream> thread_stream;
// Thread-local stream index for mapping worker threads to streams
static thread_local size_t thread_stream_idx = 0;

// Mutex protecting access to the jobs map
static std::mutex jobs_mutex;
// Global map of job_id → JobState, tracking async job progress
static std::map<int, std::unique_ptr<JobState>> jobs;

struct PinnedBufferInfo {
    void* ptr = nullptr;
    size_t size = 0;
};

static std::vector<PinnedBufferInfo> g_pinned_buffers;
void preallocate_pinned_buffers(size_t io_threads, size_t pinned_buffer_size_mb) {
    g_pinned_buffers.resize(io_threads);
    size_t alloc_bytes = pinned_buffer_size_mb * 1024 * 1024;

    for (size_t i = 0; i < io_threads; ++i) {
        void* ptr = nullptr;
        hipError_t err = hipHostMalloc(&ptr, alloc_bytes);
        if (err != hipSuccess) {
            std::cerr << "[ERROR] Failed to allocate pinned buffer for thread "
                      << i << ": " << hipGetErrorString(err) << std::endl;
            g_pinned_buffers[i] = {nullptr, 0};
        } else {
            g_pinned_buffers[i] = {ptr, alloc_bytes};

        }
    }

    std::cout << "[INFO] Pre-allocated pinned buffer "
            << (alloc_bytes / (1024 * 1024))
            << " MB for " << io_threads << " threads" << std::endl;
}

// -------------------------------
// I/O thread pool for scheduling
// -------------------------------
class IOThreadPool {
private:
    std::vector<std::thread> workers;
    std::queue<std::function<void()>> tasks;
    std::mutex queue_mutex;
    std::condition_variable condition;
    std::atomic<bool> stop{false};

public:
    IOThreadPool(int threads, size_t pinned_buffer_mb, int tp_rank) {
        // Initialize PyTorch threading globally (main thread only)
        at::init_num_threads();
        at::set_num_threads(1);

        for (size_t i = 0; i < threads; ++i) {
            workers.emplace_back([this, i, threads, pinned_buffer_mb, tp_rank] {
                // Compute unique CPU per thread
                int cpu_id = tp_rank * threads + i;
                cpu_set_t cpuset;
                CPU_ZERO(&cpuset);
                CPU_SET(cpu_id, &cpuset);

                if (pthread_setaffinity_np(pthread_self(), sizeof(cpuset), &cpuset) != 0) {
                    std::cerr << "[WARN] Failed to set affinity for thread " << i
                              << " tp_rank=" << tp_rank << " to CPU " << cpu_id << "\n";
                }

                pid_t tid = static_cast<pid_t>(syscall(SYS_gettid));
                int actual_cpu = sched_getcpu();

                std::cout << "[INFO] IO thread " << i
                          << " (tid=" << tid << ", tp_rank=" << tp_rank
                          << ") pinned to CPU " << cpu_id
                          << " (running on CPU " << actual_cpu << ")\n";

                if (i < g_pinned_buffers.size() && g_pinned_buffers[i].ptr != nullptr) {
                    t_pinned_ptr = g_pinned_buffers[i].ptr;
                    t_pinned_size = g_pinned_buffers[i].size;
                    std::cout << "[INFO] IO thread " << i
                            << " attached to preallocated pinned buffer "
                            << (t_pinned_size / (1024 * 1024)) << " MB\n";
                } else {
                    std::cerr << "[WARN] IO thread " << i
                            << " has no preallocated pinned buffer\n";
                }

                thread_stream_idx = i;
                while (true) {
                    std::function<void()> task;
                    {
                        std::unique_lock<std::mutex> lock(queue_mutex);
                        condition.wait(lock, [this] { return stop || !tasks.empty(); });
                        if (stop && tasks.empty()) return;
                        task = std::move(tasks.front());
                        tasks.pop();
                    }
                    task();
                }
            });
        }
    }

    ~IOThreadPool() {
        stop = true;
        condition.notify_all();
        for (std::thread &worker : workers) {
            worker.join();
        }
    }

    template<class F>
    auto enqueue(F&& f) -> std::future<typename std::result_of<F()>::type> {
        using return_type = typename std::result_of<F()>::type;

        auto task = std::make_shared<std::packaged_task<return_type()>>(
            std::forward<F>(f)
        );

        std::future<return_type> res = task->get_future();
        {
            std::unique_lock<std::mutex> lock(queue_mutex);
            if (stop) throw std::runtime_error("enqueue on stopped ThreadPool");
            tasks.emplace([task](){ (*task)(); });
        }
        condition.notify_one();
        return res;
    }
};

// Global IO thread pool for scheduling async PUT/GET tasks
static std::unique_ptr<IOThreadPool> g_io_pool;

// -------------------------------
// Initialize resources with pre-allocation
// -------------------------------

// Initialize IO threads, CUDA streams, and pinned memory pool
void init_performance_resources(int io_threads = 0, size_t pinned_buffer_size_mb = 256 ,size_t max_pinned_memory_gb = 10, int tp_rank = 1) {
    if (!g_io_pool) {
        if (io_threads == 0) {
            io_threads = std::max(4u, std::thread::hardware_concurrency() / 2);
        }
        std::cout << "[INFO] Initializing IOThreadPool with "
                  << io_threads << " threads, "
                  << pinned_buffer_size_mb << " MB pinned buffer per thread, "
                  << max_pinned_memory_gb << " GB max pinned memory\n"
                  << std::thread::hardware_concurrency() << " hardware_concurrency\n";

        // Pre-allocate pinned buffers before launching threads
        preallocate_pinned_buffers(io_threads, pinned_buffer_size_mb);

        g_io_pool = std::make_unique<IOThreadPool>(io_threads, pinned_buffer_size_mb, tp_rank);
        // Create dedicated streams for each thread
        g_streams.clear();
        g_streams.reserve(io_threads);
        for (size_t i = 0; i < io_threads; i++) {
            g_streams.push_back(at::cuda::getStreamFromPool(/*isHighPriority=*/false));
        }

        // Warm up CUDA context and streams
        for (auto& stream : g_streams) {
            hipStreamSynchronize(stream.stream());
        }
    }
}


// -------------------------------
// Status and cleanup
// -------------------------------
// Return finished jobs and their success status
std::vector<std::pair<int, bool>> get_finished_ext() {
    std::lock_guard<std::mutex> lock(jobs_mutex);

    std::vector<std::pair<int, bool>> results;
    std::vector<int> to_erase;

    // Iterate over all active jobs.
    for (auto &kv : jobs) {
        int job_id = kv.first;
        auto &job_state = kv.second;

        // Check if the job has completed all its tasks.
        if (job_state->completed_tasks.load() == job_state->total_tasks.load()) {
            bool all_ok = job_state->all_success.load();
            results.emplace_back(job_id, all_ok);
            to_erase.push_back(job_id);
        }
    }

    // Remove all finished jobs from the map.
    for (int jid : to_erase) {
        jobs.erase(jid);
    }
    return results;
}

// Release IO threads, CUDA streams, and pinned buffer
void cleanup_performance_resources() {
    g_io_pool.reset();
    g_streams.clear();

    if (t_pinned_ptr) {
        hipHostFree(t_pinned_ptr);
        t_pinned_ptr = nullptr;
        t_pinned_size = 0;
    }
}

//----------------------------------------------------------------------
// GPU → Storage (PUT)
// ----------------------------------------------------------------------
// Copy selected GPU tensor blocks into pinned CPU buffer asynchronously
torch::Tensor copy_gpu_tensors_to_buffer_async(
    const std::vector<torch::Tensor>& src_tensors,
    const std::vector<int64_t>& block_ids_list,
    const c10::cuda::CUDAStream& stream) {

    // Calculate total size first
    size_t total_elements = 0;
    auto dtype = src_tensors[0].dtype();
    size_t element_size = src_tensors[0].element_size();

    for (int64_t block_id : block_ids_list) {
        for (const auto &tensor : src_tensors) {
            auto block = tensor.index({torch::indexing::Slice(), block_id});
            total_elements += block.numel();
        }
    }

    size_t required_bytes = total_elements * element_size;
    auto [pinned_ptr, pinned_size] = get_thread_local_pinned(required_bytes);

    torch::Tensor result_cpu;

    // Use pinned memory - much faster transfer
    std::vector<torch::Tensor> blocks;
    blocks.reserve(src_tensors.size() * block_ids_list.size());

    for (int64_t block_id : block_ids_list) {
        for (const auto &tensor : src_tensors) {
            auto block = tensor.index({torch::indexing::Slice(), block_id});
            blocks.push_back(block.contiguous());
        }
    }

    // Concatenate all selected [K,V] block slices into one big tensor along dim=0 → [2 * num_layers * num_blocks, H, B, D]
    auto flat_gpu = torch::cat(blocks, 0);

    // Create memory view of CPU tensor using pinned memory
    result_cpu = torch::from_blob(
        pinned_ptr,
        {static_cast<long>(total_elements)},
        torch::TensorOptions().dtype(dtype).device(torch::kCPU).pinned_memory(true)
    );

    // Async copy GPU -> pinned CPU memory
    hipMemcpyAsync(
        pinned_ptr,
        flat_gpu.data_ptr(),
        required_bytes,
        hipMemcpyDeviceToHost,
        stream.stream()
    );

    if (result_cpu.dtype() == torch::kBFloat16) {
        result_cpu = result_cpu.view(torch::kUInt16);
    }

    return result_cpu;
}


// File write helpers - optimized for large writes
bool write_file_to_disk(const std::string &target_path,
                            const torch::Tensor &host_buf) {
    const void* data_ptr = host_buf.data_ptr();
    // Get total number of bytes to write
    size_t nbytes = host_buf.nbytes();
    // Write first to a temporary file to ensure atomic rename later
    std::string tmp_path = target_path + ".tmp";

    // Define a larger buffer (1MB) to reduce syscall overhead and speed up I/O
    const size_t WRITE_BUFFER_SIZE = 1 * 1024 * 1024; // 1MB buffer

    std::ofstream ofs(tmp_path, std::ios::out | std::ios::binary);
    if (!ofs) {
        std::cerr << "[ERROR] Failed to open temporary file for writing: " << tmp_path << "\n";
        return false;
    }

    // Allocate custom I/O buffer for this stream (replaces small default buffer)
    std::vector<char> buffer(WRITE_BUFFER_SIZE);
    // Apply the custom buffer to the file stream
    ofs.rdbuf()->pubsetbuf(buffer.data(), WRITE_BUFFER_SIZE);

    ofs.write(reinterpret_cast<const char*>(data_ptr), nbytes);
    ofs.close();
    // std::cout << "[INFO] Written " << nbytes << " bytes to temporary file: " << tmp_path << "\n";
    // Atomically rename temp file to final target name after successful write
    return (std::rename(tmp_path.c_str(), target_path.c_str()) == 0);
}


// Async GPU → Storage transfer (PUT)
bool transfer_async_put_ext(int job_id,
                        std::vector<std::string> target_files,
                        std::vector<torch::Tensor> src_tensors,
                        std::vector<std::vector<int64_t>> all_block_ids) {

    // Create job state object that will track progress and futures for this job.
    auto job_state = std::make_unique<JobState>();
    job_state->total_tasks = target_files.size();

    // For each target file, enqueue one async task in the I/O thread pool.
    for (size_t i = 0; i < target_files.size(); i++) {
        std::string target = target_files[i];
        auto src = src_tensors;
        auto bids = all_block_ids[i];

        auto future = g_io_pool->enqueue([=, job_state = job_state.get()]() -> bool {

          // Each thread gets a dedicated CUDA stream for async GPU ops.
            if (!thread_stream.has_value()) {
                thread_stream = g_streams[thread_stream_idx % g_streams.size()];
            }

            // Save current CUDA stream so we can restore it later.
            auto current_stream = at::cuda::getCurrentCUDAStream();
            at::cuda::setCurrentCUDAStream(*thread_stream);

            try {
                // Stage 1: Asynchronously copy tensors from GPU to pinned CPU buffer.
                auto host_buf = copy_gpu_tensors_to_buffer_async(src, bids, *thread_stream);

                // Stage 2: Synchronize only this thread's CUDA stream (not all).
                hipStreamSynchronize(thread_stream->stream());

                // Stage 3: Write the pinned buffer to disk (blocking operation).
                bool ok = TIME_EXPR("write_file_to_disk", write_file_to_disk(target, host_buf));
                //bool ok = write_file_to_disk(target, host_buf);

                if (!ok)
                    std::cerr << "[ERROR] PUT failed during file write: " << target << "\n";

                // Restore original CUDA stream for safety.
                at::cuda::setCurrentCUDAStream(current_stream);

                // Atomically mark task completion.
                job_state->completed_tasks.fetch_add(1);
                if (!ok) job_state->all_success = false;
                return ok;

            } catch (...) {
                at::cuda::setCurrentCUDAStream(current_stream);
                job_state->completed_tasks.fetch_add(1);
                job_state->all_success = false;
                std::cerr << "[ERROR] PUT failed for " << target << "\n";
                return false;
            }
        });

        // Convert std::future → std::shared_future- is copyable and can be waited on by multiple threads.
        job_state->futures.push_back(future.share());
    }

    std::lock_guard<std::mutex> lock(jobs_mutex); // protect jobs map
    jobs[job_id] = std::move(job_state);

    return true;
}
// ----------------------------------------------------------------------
// Storage -> GPU (GET)
// ----------------------------------------------------------------------

// Read a file into a pinned CPU tensor from the pool
torch::Tensor read_file_to_pinned_tensor(const std::string& path) {

    // Open file
    std::ifstream ifs(path, std::ios::in | std::ios::binary | std::ios::ate);
    if (!ifs) {
        std::cerr << "[ERROR] Failed to open file: " << path << "\n";
        throw std::runtime_error("Failed to open file: " + path);
    }

    // Determine file size
    size_t file_size = static_cast<size_t>(ifs.tellg());
    ifs.seekg(0, std::ios::beg);

    // Get pinned buffer from pool
    auto [pinned_ptr, pinned_size] = get_thread_local_pinned(file_size);
    if (!pinned_ptr || pinned_size < file_size) {
        std::cerr << "[ERROR] Pinned buffer too small for file: " << path << "\n"
                  << "[INFO] Required size: " << file_size << " bytes, Available size: " << pinned_size << " bytes\n"
                  << "pinned_ptr: " << pinned_ptr << "\n";
        throw std::runtime_error("Pinned buffer too small for file: " + path);
    }

    // Read into pinned memory
    ifs.read(reinterpret_cast<char*>(pinned_ptr), file_size);
    ifs.close();
    //std::cout << "[INFO] 1/2 Read file into pinned tensor: " << path << " (" << file_size << " bytes)\n";
    // Wrap pinned buffer into a Torch tensor (CPU, pinned)
    auto options = torch::TensorOptions()
        .dtype(torch::kUInt8)  // raw bytes
        .device(torch::kCPU)
        .pinned_memory(true);

    // Wrap without copy, need free pinned_ptr.
    auto tensor = torch::from_blob(
        pinned_ptr,
        {static_cast<long>(file_size)},
        [pinned_ptr](void* /*unused*/) {},
        options
    );
    // std::cout << "[INFO] 2/2 Read file into pinned tensor: " << path << " (" << file_size << " bytes)\n";
    return tensor;
}
// Copy buffer to GPU tensors asynchronously using hipMemcpyAsync (no swap_blocks)
// CPU layout (contiguous): [num_blocks_in_file, num_layers, 2, H, B, D]
// GPU layout (contiguous): [2, num_blocks_total, H, B, D]
bool copy_buffer_to_gpu_tensors(
    torch::Tensor cpu_buf,
    const std::vector<int64_t>& block_ids_list,
    const std::vector<torch::Tensor>& dst_tensors,
    int num_blocks_in_file,
    const c10::cuda::CUDAStream& stream) {

    TORCH_CHECK(!dst_tensors.empty(), "Destination tensors list is empty");
    const auto& ref = dst_tensors[0];
    TORCH_CHECK(ref.is_contiguous(), "dst_tensors must be contiguous");
    TORCH_CHECK(cpu_buf.is_contiguous(), "cpu buffer must be contiguous");

    const auto shape = ref.sizes();  // [2, num_blocks_total, H, B, D]
    TORCH_CHECK(shape.size() == 5, "Expected shape [2, num_blocks, H, B, D]");

    const int64_t num_blocks_tot = shape[1];
    const int64_t H = shape[2];
    const int64_t B = shape[3];
    const int64_t D = shape[4];
    const int num_layers = static_cast<int>(dst_tensors.size());

    const size_t elem_size = ref.element_size();
    const size_t bytes_per_plane = static_cast<size_t>(H * B * D) * elem_size;
    const size_t bytes_per_block = 2 * bytes_per_plane;  // [K,V] in CPU buf

    auto* cpu_base = cpu_buf.data_ptr<uint8_t>();

    for (size_t bi = 0; bi < block_ids_list.size(); ++bi) {
        const int64_t gblock = block_ids_list[bi];
        const int64_t lblock = (num_blocks_in_file > 0) ? (gblock % num_blocks_in_file) : 0;
        const size_t cpu_block_base = static_cast<size_t>(lblock) * num_layers * bytes_per_block;

        for (int layer = 0; layer < num_layers; ++layer) {
            const auto& layer_tensor = dst_tensors[layer];
            auto* dst_base = reinterpret_cast<uint8_t*>(layer_tensor.data_ptr());

            // Compute CPU source offsets for K and V
            const size_t cpu_K_off = cpu_block_base + static_cast<size_t>(layer) * bytes_per_block;
            const size_t cpu_V_off = cpu_K_off + bytes_per_plane;

            // Compute GPU destination offsets for K and V
            const size_t gpu_K_off = static_cast<size_t>(gblock) * bytes_per_plane;
            const size_t gpu_V_off = (static_cast<size_t>(num_blocks_tot) + gblock) * bytes_per_plane;

            void* dst_K = dst_base + gpu_K_off;
            void* dst_V = dst_base + gpu_V_off;
            void* src_K = cpu_base + cpu_K_off;
            void* src_V = cpu_base + cpu_V_off;


            hipError_t err1 = hipMemcpyAsync(
                dst_K, src_K, bytes_per_plane,
                hipMemcpyHostToDevice, stream.stream());

            hipError_t err2 = hipMemcpyAsync(
                dst_V, src_V, bytes_per_plane,
                hipMemcpyHostToDevice, stream.stream());

            if (err1 != hipSuccess || err2 != hipSuccess) {
                std::cerr << "[ERROR] hipMemcpyAsync failed for block=" << gblock
                          << " layer=" << layer
                          << " err1=" << hipGetErrorString(err1)
                          << " err2=" << hipGetErrorString(err2)
                          << std::endl;
                return false;
            }
        }
    }

    return true;
}

// Async Storage → GPU transfer (GET)
bool transfer_async_get_ext(
    int job_id,
    std::vector<std::string> source_files,
    std::vector<std::vector<int64_t>> all_block_ids,
    std::vector<torch::Tensor> dst_tensors,
    int gpu_blocks_per_file)
{
    // Create job state object to track progress and futures for this job.
    auto job_state = std::make_unique<JobState>();
    job_state->total_tasks = source_files.size();

    // For each source file, enqueue one async task in the I/O thread pool.
    for (size_t i = 0; i < source_files.size(); i++) {
        std::string src_file = source_files[i];
        auto block_ids = all_block_ids[i];
        auto future = g_io_pool->enqueue([=, job_state = job_state.get()]() -> bool {

            // Get dedicated stream for this thread
            if (!thread_stream.has_value()) {
                thread_stream = g_streams[thread_stream_idx % g_streams.size()];
            }

            // Save current CUDA stream so we can restore it later.
            auto current_stream = at::cuda::getCurrentCUDAStream();
            at::cuda::setCurrentCUDAStream(*thread_stream);

            // -------------------------
            // Stage 1: File → pinned CPU
            // -------------------------
            bool stage1_ok = false;
            torch::Tensor host_buf;
            try {
                // Read data from disk into pinned memory tensor.
                //host_buf = read_file_to_pinned_tensor(src_file);
                host_buf = TIME_EXPR("read_file_to_pinned_tensor", read_file_to_pinned_tensor(src_file));
                stage1_ok = true;
            } catch (const std::exception& e) {
                std::cerr << "[ERROR] Stage1 read_file_to_pinned_tensor failed for "
                        << src_file << ": " << e.what() << std::endl;
            } catch (...) {
                std::cerr << "[ERROR] Stage1 unknown failure for " << src_file << std::endl;
            }

            // -------------------------
            // Stage 2: CPU → GPU copy
            // -------------------------
            bool ok = false;
            if (stage1_ok) {
                try {
                    // Perform asynchronous GPU copy and tensor swap.
                    ok = copy_buffer_to_gpu_tensors(
                        host_buf, block_ids, dst_tensors, gpu_blocks_per_file, *thread_stream);

                    hipError_t err = hipStreamSynchronize(thread_stream->stream());
                    if (err != hipSuccess) {
                        std::cerr << "[ERROR] hipStreamSynchronize failed: "
                                << hipGetErrorString(err) << std::endl;
                        ok = false;
                    }

                } catch (const std::exception& e) {
                    std::cerr << "[ERROR] Stage2 copy_and_swap failed for " << src_file
                            << ": " << e.what() << std::endl;
                } catch (...) {
                    std::cerr << "[ERROR] Stage2 unknown failure for " << src_file << std::endl;
                }
            }

            // -------------------------
            // Final cleanup & accounting
            // -------------------------
            // Synchronize only this thread's CUDA stream.
            at::cuda::setCurrentCUDAStream(current_stream);
            job_state->completed_tasks.fetch_add(1);
            if (!ok) job_state->all_success = false;
            return ok;
        });
        // Convert std::future → std::shared_future- is copyable and can be waited on by multiple threads.

        job_state->futures.push_back(future.share());
    }

    std::lock_guard<std::mutex> lock(jobs_mutex);
    jobs[job_id] = std::move(job_state);
    return true;
}

// -------------------------------
// PYBIND11 module
// -------------------------------
PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("init_performance_resources", &init_performance_resources,
          py::arg("io_threads") = 0,
          py::arg("pinned_buffer_size_mb") = 256,
          py::arg("max_pinned_memory_gb") = 50,
          py::arg("tp_rank") = 1);

    m.def("cleanup_performance_resources", &cleanup_performance_resources);

    m.def("get_finished_ext", &get_finished_ext);

    m.def("transfer_async_put_ext", &transfer_async_put_ext,
          "Async transfer  GPU-> Storage",
          py::arg("job_id"),
          py::arg("target_files"),
          py::arg("all_src_tensors"),
          py::arg("all_block_ids"));

    m.def("transfer_async_get_ext", &transfer_async_get_ext,
      "Async GET transfer from Storage → GPU",
      py::arg("job_id"),
      py::arg("source_files"),
      py::arg("all_block_ids"),
      py::arg("dst_tensors"),
      py::arg("gpu_blocks_per_file"));
}
