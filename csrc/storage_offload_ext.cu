#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <pybind11/pybind11.h>
#include <c10/cuda/CUDAStream.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>

#include <iostream>
#include <algorithm>
#include <cassert>
#include <fstream>
#include <string>
#include <vector>
#include <chrono>
#include <map>
#include <future>
#include <mutex>
#include <thread>
#include <queue>
#include <condition_variable>
#include <memory>
#include <atomic>
#include <optional>

namespace py = pybind11;

// -------------------------------------
// Thread-local pinned buffer
// -------------------------------------
static thread_local void* t_pinned_ptr = nullptr;
static thread_local size_t t_pinned_size = 0;

static std::pair<void*, size_t> get_thread_local_pinned(size_t required_bytes) {
    if (!t_pinned_ptr || t_pinned_size < required_bytes) {
        if (t_pinned_ptr) {
            hipHostFree(t_pinned_ptr);
            t_pinned_ptr = nullptr;
            t_pinned_size = 0;
        }

        size_t alloc_size = std::max(required_bytes, (size_t)16 * 1024 * 1024); // at least 16 MB
        hipError_t err = hipHostMalloc(&t_pinned_ptr, alloc_size);
        if (err != hipSuccess) {
            std::cerr << "[ERROR] hipHostMalloc failed: "
                      << hipGetErrorString(err) << "\n";
            t_pinned_ptr = nullptr;
            t_pinned_size = 0;
        } else {
            t_pinned_size = alloc_size;
            std::cout << "[INFO] Thread " << std::this_thread::get_id()
                      << " allocated pinned buffer "
                      << (alloc_size / (1024 * 1024)) << " MB\n";
        }
    }
    return {t_pinned_ptr, t_pinned_size};
}


// Tracks async job progress and results
struct JobState {
    // Futures for each async task in the job
    std::vector<std::shared_future<bool>> futures;
    // Number of tasks completed so far
    std::atomic<int> completed_tasks{0};
    // Total number of tasks scheduled for this job
    std::atomic<int> total_tasks{0};
    // Flag indicating if all tasks succeeded
    std::atomic<bool> all_success{true};
};
// --------------------------------------
// Global resources - TODO should removed all Global variables
// --------------------------------------
// CUDA streams assigned to each worker thread
static std::vector<at::cuda::CUDAStream> g_streams;

// Thread-local CUDA stream used by the current worker
static thread_local std::optional<c10::cuda::CUDAStream> thread_stream;
// Thread-local stream index for mapping worker threads to streams
static thread_local size_t thread_stream_idx = 0;

// Mutex protecting access to the jobs map
static std::mutex jobs_mutex;
// Global map of job_id → JobState, tracking async job progress
static std::map<int, std::unique_ptr<JobState>> jobs;

// -------------------------------
// I/O thread pool for scheduling
// -------------------------------
class IOThreadPool {
private:
    std::vector<std::thread> workers;
    std::queue<std::function<void()>> tasks;
    std::mutex queue_mutex;
    std::condition_variable condition;
    std::atomic<bool> stop{false};

public:
    IOThreadPool(size_t threads, size_t pinned_buffer_mb = 16) {
        // Initialize PyTorch threading globally (main thread only)
        at::init_num_threads();
        at::set_num_threads(1);

        for (size_t i = 0; i < threads; ++i) {
            workers.emplace_back([this, i, pinned_buffer_mb] {
                // Each I/O thread preallocates its own pinned buffer
                size_t alloc_bytes = pinned_buffer_mb * 1024 * 1024;
                auto [ptr, size] = get_thread_local_pinned(alloc_bytes);

                if (ptr) {
                    std::cout << "[INFO] IO thread " << i
                              << " preallocated pinned buffer "
                              << (size / (1024 * 1024)) << " MB\n";
                } else {
                    std::cerr << "[WARN] IO thread " << i
                              << " failed to allocate pinned buffer\n";
                }

                // Set thread affinity to prevent migration overhead
                thread_stream_idx = i;
                while (true) {
                    std::function<void()> task;
                    {
                        std::unique_lock<std::mutex> lock(queue_mutex);
                        condition.wait(lock, [this] { return stop || !tasks.empty(); });
                        if (stop && tasks.empty()) return;
                        task = std::move(tasks.front());
                        tasks.pop();
                    }
                    task();
                }
            });
        }
    }

    ~IOThreadPool() {
        stop = true;
        condition.notify_all();
        for (std::thread &worker : workers) {
            worker.join();
        }
    }

    template<class F>
    auto enqueue(F&& f) -> std::future<typename std::result_of<F()>::type> {
        using return_type = typename std::result_of<F()>::type;

        auto task = std::make_shared<std::packaged_task<return_type()>>(
            std::forward<F>(f)
        );

        std::future<return_type> res = task->get_future();
        {
            std::unique_lock<std::mutex> lock(queue_mutex);
            if (stop) throw std::runtime_error("enqueue on stopped ThreadPool");
            tasks.emplace([task](){ (*task)(); });
        }
        condition.notify_one();
        return res;
    }
};

// Global IO thread pool for scheduling async PUT/GET tasks
static std::unique_ptr<IOThreadPool> g_io_pool;

// -------------------------------
// Initialize resources with pre-allocation
// -------------------------------

// Initialize IO threads, CUDA streams, and pinned memory pool
void init_performance_resources(size_t io_threads = 0, size_t pinned_buffer_size_mb = 256 ,size_t max_pinned_memory_gb = 10) {
    if (!g_io_pool) {
        if (io_threads == 0) {
            io_threads = std::max(4u, std::thread::hardware_concurrency() / 2);
        }
        std::cout << "[INFO] Initializing IOThreadPool with "
                  << io_threads << " threads, "
                  << pinned_buffer_size_mb << " MB pinned buffer per thread, "
                  << max_pinned_memory_gb << " GB max pinned memory\n"
                  << std::thread::hardware_concurrency() << " hardware_concurrency\n";

        g_io_pool = std::make_unique<IOThreadPool>(io_threads, pinned_buffer_size_mb);

        // Create dedicated streams for each thread
        g_streams.clear();
        g_streams.reserve(io_threads);
        for (size_t i = 0; i < io_threads; i++) {
            g_streams.push_back(at::cuda::getStreamFromPool(/*isHighPriority=*/false));
        }

        // Pre-allocate pinned memory pool
        size_t buffer_size = pinned_buffer_size_mb * 1024 * 1024;
        size_t max_buffers = std::min(io_threads, (max_pinned_memory_gb * 1024 * 1024 * 1024) / buffer_size);

        if (max_buffers == 0) {
           throw std::runtime_error("PinnedMemoryPool size is zero. Increase max_pinned_memory_gb or decrease pinned_buffer_size_mb.");
        }
        // Warm up CUDA context and streams
        for (auto& stream : g_streams) {
            hipStreamSynchronize(stream.stream());
        }
    }
}

// -------------------------------
// Optimized GPU → CPU copy with zero-copy when possible
// -------------------------------
// Copy selected GPU tensor blocks into pinned CPU buffer asynchronously
torch::Tensor copy_gpu_tensors_to_buffer_async(
    const std::vector<torch::Tensor>& src_tensors,
    const std::vector<int64_t>& block_ids_list,
    const c10::cuda::CUDAStream& stream) {

    // Calculate total size first
    size_t total_elements = 0;
    auto dtype = src_tensors[0].dtype();
    size_t element_size = src_tensors[0].element_size();

    for (int64_t block_id : block_ids_list) {
        for (const auto &tensor : src_tensors) {
            auto block = tensor.index({torch::indexing::Slice(), block_id});
            total_elements += block.numel();
        }
    }


    size_t required_bytes = total_elements * element_size;
    auto [pinned_ptr, pinned_size] = get_thread_local_pinned(required_bytes);

    torch::Tensor result_cpu;

    if (pinned_ptr && pinned_size >= required_bytes) {
        // Use pinned memory - much faster transfer
        std::vector<torch::Tensor> blocks;
        blocks.reserve(src_tensors.size() * block_ids_list.size());

        for (int64_t block_id : block_ids_list) {
            for (const auto &tensor : src_tensors) {
                auto block = tensor.index({torch::indexing::Slice(), block_id});
                blocks.push_back(block.contiguous());
            }
        }

        // Concatenate all selected [K,V] block slices into one big tensor along dim=0 → [2 * num_layers * num_blocks, H, B, D]
        auto flat_gpu = torch::cat(blocks, 0);

        // Create memory view of CPU tensor using pinned memory
        result_cpu = torch::from_blob(
            pinned_ptr,
            {static_cast<long>(total_elements)},
            torch::TensorOptions().dtype(dtype).device(torch::kCPU).pinned_memory(true)
        );

        // Async copy GPU -> pinned CPU memory
        hipMemcpyAsync(
            pinned_ptr,
            flat_gpu.data_ptr(),
            required_bytes,
            hipMemcpyDeviceToHost,
            stream.stream()
        );

    } else {
        // Fallback to regular memory

        std::vector<torch::Tensor> blocks;
        blocks.reserve(src_tensors.size() * block_ids_list.size());

        for (int64_t block_id : block_ids_list) {
            for (const auto &tensor : src_tensors) {
                auto block = tensor.index({torch::indexing::Slice(), block_id});
                blocks.push_back(block);
            }
        }

        auto flat = torch::cat(blocks, 0);
        result_cpu = flat.contiguous().to(torch::kCPU, /*non_blocking=*/true);
    }

    if (result_cpu.dtype() == torch::kBFloat16) {
        result_cpu = result_cpu.view(torch::kUInt16);
    }

    return result_cpu;
}

// -------------------------------
// Status and cleanup
// -------------------------------
// Return finished jobs and their success status
std::vector<std::pair<int, bool>> get_finished_ext() {
    std::lock_guard<std::mutex> lock(jobs_mutex);

    std::vector<std::pair<int, bool>> results;
    std::vector<int> to_erase;

    // Iterate over all active jobs.
    for (auto &kv : jobs) {
        int job_id = kv.first;
        auto &job_state = kv.second;

        // Check if the job has completed all its tasks.
        if (job_state->completed_tasks.load() == job_state->total_tasks.load()) {
            bool all_ok = job_state->all_success.load();
            results.emplace_back(job_id, all_ok);
            to_erase.push_back(job_id);
        }
    }

    // Remove all finished jobs from the map.
    for (int jid : to_erase) {
        jobs.erase(jid);
    }
    return results;
}

// Release IO threads, CUDA streams, and pinned pool
void cleanup_performance_resources() {
    g_io_pool.reset();
    g_streams.clear();

    if (t_pinned_ptr) {
        hipHostFree(t_pinned_ptr);
        t_pinned_ptr = nullptr;
        t_pinned_size = 0;
    }
}

//----------------------------------------------------------------------
// GPU → Storage (PUT)
// ----------------------------------------------------------------------

// File write helpers - optimized for large writes
bool flush_one_to_disk_fast(const std::string &target_path,
                            const torch::Tensor &host_buf) {
    const void* data_ptr = host_buf.data_ptr();
    size_t nbytes = host_buf.nbytes();
    std::string tmp_path = target_path + ".tmp";

    // Use larger buffer for faster I/O
    const size_t WRITE_BUFFER_SIZE = 1024 * 1024; // 1MB buffer

    std::ofstream ofs(tmp_path, std::ios::out | std::ios::binary);
    if (!ofs) {
        std::cerr << "[ERROR] Failed to open temporary file for writing: " << tmp_path << "\n";
        return false;
    }

    // Set larger buffer
    std::vector<char> buffer(WRITE_BUFFER_SIZE);
    ofs.rdbuf()->pubsetbuf(buffer.data(), WRITE_BUFFER_SIZE);

    ofs.write(reinterpret_cast<const char*>(data_ptr), nbytes);
    ofs.close();
    // std::cout << "[INFO] Written " << nbytes << " bytes to temporary file: " << tmp_path << "\n";
    return (std::rename(tmp_path.c_str(), target_path.c_str()) == 0);
}


// Async GPU → Storage transfer (PUT)
bool transfer_async_put_ext(int job_id,
                        std::vector<std::string> target_files,
                        std::vector<torch::Tensor> src_tensors,
                        std::vector<std::vector<int64_t>> all_block_ids) {
    init_performance_resources();

    std::lock_guard<std::mutex> lock(jobs_mutex);
    auto job_state = std::make_unique<JobState>();
    job_state->total_tasks = target_files.size();

    for (size_t i = 0; i < target_files.size(); i++) {
        std::string target = target_files[i];
        auto src = src_tensors;
        auto bids = all_block_ids[i];

        auto future = g_io_pool->enqueue([=, job_state = job_state.get()]() -> bool {

            // Get dedicated stream for this thread
            if (!thread_stream.has_value()) {
                thread_stream = g_streams[thread_stream_idx % g_streams.size()];
            }

            auto current_stream = at::cuda::getCurrentCUDAStream();
            at::cuda::setCurrentCUDAStream(*thread_stream);

            try {
                // Stage 1: Start async GPU → CPU copy (non-blocking)
                auto host_buf = copy_gpu_tensors_to_buffer_async(src, bids, *thread_stream);

                // Stage 2: Only synchronize the specific stream we need
                hipStreamSynchronize(thread_stream->stream());

                // Stage 3: Write to disk (this is now the only blocking operation)
                bool ok = flush_one_to_disk_fast(target, host_buf);
                if (!ok) {
                    std::cerr << "[ERROR] PUT failed during file write: " << target << "\n";
                }
                //  else {
                //     std::cout << "[INFO] Put to file completed: " << target << "\n";
                // }

                at::cuda::setCurrentCUDAStream(current_stream);

                job_state->completed_tasks.fetch_add(1);
                if (!ok) job_state->all_success = false;
                return ok;

            } catch (...) {
                at::cuda::setCurrentCUDAStream(current_stream);
                job_state->completed_tasks.fetch_add(1);
                job_state->all_success = false;
                std::cerr << "[ERROR] PUT failed for " << target << "\n";
                return false;
            }
        });

        job_state->futures.push_back(future.share());
    }

    jobs[job_id] = std::move(job_state);
    return true;
}



// ----------------------------------------------------------------------
// Storage -> GPU (GET)
// ----------------------------------------------------------------------
// Swap KV blocks between source and destination tensors (multi-layer)
void swap_blocks_multi_layer(
    const std::vector<torch::Tensor>& src_kv_caches,
    const std::vector<torch::Tensor>& dst_kv_caches,
    const torch::Tensor& block_mapping) {

  TORCH_CHECK(src_kv_caches.size() == dst_kv_caches.size(),
              "src and dst must have the same number of layers");
  TORCH_CHECK(block_mapping.device().is_cpu(),
              "block_mapping must be on CPU");

  // Assume all tensors are on same device
  torch::Device src_device = src_kv_caches[0].device();
  torch::Device dst_device = dst_kv_caches[0].device();

  hipMemcpyKind memcpy_type;
  if (src_device.is_cuda() && dst_device.is_cuda()) {
    TORCH_CHECK(src_device.index() == dst_device.index(),
                "src and dst must be on the same GPU");
    memcpy_type = hipMemcpyDeviceToDevice;
  } else if (src_device.is_cuda() && dst_device.is_cpu()) {
    memcpy_type = hipMemcpyDeviceToHost;
  } else if (src_device.is_cpu() && dst_device.is_cuda()) {
    memcpy_type = hipMemcpyHostToDevice;
  } else {
    TORCH_CHECK(false, "Invalid device combination");
  }

  const int64_t num_blocks = block_mapping.size(0);
  const at::cuda::OptionalCUDAGuard device_guard(
      src_device.is_cuda() ? src_device : dst_device);
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();

  // Loop over layers
  for (size_t layer = 0; layer < src_kv_caches.size(); ++layer) {
    auto src = src_kv_caches[layer];
    auto dst = dst_kv_caches[layer];

    TORCH_CHECK(src.dim() >= 2 && dst.dim() >= 2,
                "Expected each KV cache tensor to have at least 2 dims "
                "(kv, blocks, ...)");
    TORCH_CHECK(src.size(0) == 2 && dst.size(0) == 2,
                "First dimension must be size=2 (key, value)");

    // For both key/value (dim 0 split)
    for (int kv = 0; kv < 2; ++kv) {
      auto src_view = src.select(0, kv);
      auto dst_view = dst.select(0, kv);

      char* src_ptr = static_cast<char*>(src_view.data_ptr());
      char* dst_ptr = static_cast<char*>(dst_view.data_ptr());

      // Size per "block" in bytes.
      // We assume blocks are indexed along dim=0 of src_view.
      int64_t block_size_in_bytes =
        src_view.element_size() * src_view.stride(0);

      for (int64_t i = 0; i < num_blocks; i++) {
        int64_t src_block_number = block_mapping[i][0].item<int64_t>();
        int64_t dst_block_number = block_mapping[i][1].item<int64_t>();
        int64_t src_offset = src_block_number * block_size_in_bytes;
        int64_t dst_offset = dst_block_number * block_size_in_bytes;

        hipMemcpyAsync(dst_ptr + dst_offset,
                        src_ptr + src_offset,
                        block_size_in_bytes,
                        memcpy_type,
                        stream);
      }
    }
  }

  // Synchronize once after all copies queued
  hipStreamSynchronize(stream);
}


// Read a file into a pinned CPU tensor from the pool
torch::Tensor read_file_to_pinned_tensor(const std::string& path) {

    // Open file
    std::ifstream ifs(path, std::ios::in | std::ios::binary | std::ios::ate);
    if (!ifs) {
        std::cerr << "[ERROR] Failed to open file: " << path << "\n";
        throw std::runtime_error("Failed to open file: " + path);
    }

    // Determine file size
    size_t file_size = static_cast<size_t>(ifs.tellg());
    ifs.seekg(0, std::ios::beg);

    // Get pinned buffer from pool
    auto [pinned_ptr, pinned_size] = get_thread_local_pinned(file_size);
    if (!pinned_ptr || pinned_size < file_size) {
        std::cerr << "[ERROR] Pinned buffer too small for file: " << path << "\n"
                  << "[INFO] Required size: " << file_size << " bytes, Available size: " << pinned_size << " bytes\n"
                  << "pinned_ptr: " << pinned_ptr << "\n";
        throw std::runtime_error("Pinned buffer too small for file: " + path);
    }

    // Read into pinned memory
    ifs.read(reinterpret_cast<char*>(pinned_ptr), file_size);
    ifs.close();
    //std::cout << "[INFO] 1/2 Read file into pinned tensor: " << path << " (" << file_size << " bytes)\n";
    // Wrap pinned buffer into a Torch tensor (CPU, pinned)
    auto options = torch::TensorOptions()
        .dtype(torch::kUInt8)  // raw bytes
        .device(torch::kCPU)
        .pinned_memory(true);

    // Wrap without copy, need free pinned_ptr.
    auto tensor = torch::from_blob(
        pinned_ptr,
        {static_cast<long>(file_size)},
        [pinned_ptr](void* /*unused*/) {},
        options
    );
    // std::cout << "[INFO] 2/2 Read file into pinned tensor: " << path << " (" << file_size << " bytes)\n";
    return tensor;
}

// Copy buffer to GPU tensors and swap blocks into place
bool copy_and_swap_gpu_tensors_ext(
    torch::Tensor buf,                           // raw CPU buffer (uint8 tensor from mmap or read)
    const std::vector<int64_t>& block_ids_list,  // block IDs to load
    const std::vector<torch::Tensor>& dst_tensors,
    int gpu_blocks_per_file) {

    // 1. Shape of one KV block [2, H, B, D]
    auto sliced = dst_tensors[0].index({torch::indexing::Slice(), 0});
    auto block_shape = sliced.sizes();  // (2, B, D)

    int64_t elems_per_block = 1;
    for (auto s : block_shape) elems_per_block *= s;

    int num_layers = dst_tensors.size();

    int num_blocks = gpu_blocks_per_file;

    int64_t expected_size = elems_per_block * num_layers * num_blocks;
    // std::cout << "[DEBUG] expected_size (elements): " << expected_size
    //           << " -> bytes: " << expected_size * dst_tensors[0].element_size() << std::endl;

    // 2. Reinterpret raw buffer
    auto dtype = dst_tensors[0].dtype();
    auto options = torch::TensorOptions().dtype(dtype).device(torch::kCPU);

    // 3. Sanity check
    bool full_block = (block_ids_list.size() % gpu_blocks_per_file == 0);
    auto data_ptr = buf.data_ptr<uint8_t>();
    int64_t num_bytes = buf.numel();
    int64_t num_elems = num_bytes / dst_tensors[0].element_size();

    // Create a memory overview of CPU tensor over existing data_ptr.
    torch::Tensor flat_tensor = torch::from_blob(
        data_ptr,
        {num_elems},
        torch::TensorOptions().dtype(dst_tensors[0].dtype()).device(torch::kCPU)
    );


    int64_t numel = flat_tensor.numel();
    int64_t got_bytes = numel * flat_tensor.element_size();
    int64_t expected_bytes = expected_size * dst_tensors[0].element_size();

    if (numel != expected_size) {
        std::cerr << "[ERROR] File size mismatch: got " << numel
                  << " elems (" << got_bytes << " bytes), expected "
                  << expected_size << " elems (" << expected_bytes << " bytes)\n";
        throw std::runtime_error(
            "File size mismatch: got " + std::to_string(numel) + " elems (" +
            std::to_string(got_bytes) + " bytes), expected " +
            std::to_string(expected_size) + " elems (" +
            std::to_string(expected_bytes) + " bytes)");
    }
    // 4. Reshape into [num_blocks, num_layers, 2, H, B, D]
    // Expect [num_blocks, num_layers, 2, B, D]
    std::vector<int64_t> new_shape = {num_blocks, num_layers};
    new_shape.insert(new_shape.end(), block_shape.begin(), block_shape.end());
    // block_shape = (2, 64, 16, 128) -> [2, 64, 16, 128]
    // new_shape = [1, 80, 2, 64, 16, 128]

    auto torch_arr = flat_tensor.view(new_shape);
    // std::cout << "[DEBUG] torch_arr reshaped to: " << torch_arr.sizes() << std::endl;

    // 5. Handle bf16 special case
    if (dst_tensors[0].dtype() == torch::kBFloat16 && torch_arr.dtype() != torch::kBFloat16) {
        torch_arr = torch_arr.view(torch::kBFloat16);
    }

    // 6. Offset calculation
    int offset = 0;
    int num_read_blocks = static_cast<int>(block_ids_list.size());

    if (!full_block) {
        offset = num_blocks - block_ids_list.size();
    }


    // 7. Build src_tensors for each layer
    // Assert that slice length matches block_ids_list
    TORCH_CHECK((num_blocks - offset) == (int)block_ids_list.size(),
                "Mismatch: slice size != block_ids_list size");

    // Build list of per-layer tensors, each shaped [2, num_blocks, H, B, D]
    std::vector<torch::Tensor> src_tensors;
    src_tensors.reserve(num_layers);

    for (int i = 0; i < num_layers; i++) {
        std::vector<torch::Tensor> blocks_for_layer;
        blocks_for_layer.reserve(block_ids_list.size());

        for (int j = 0; j < (int)block_ids_list.size(); j++) {
            int global_b = block_ids_list[j];   // global block id
            int local_b  = global_b % num_blocks; // offset inside this file

            auto block_tensor = torch_arr[local_b][i];  // one block for this layer [2, H, B, D]
            blocks_for_layer.push_back(block_tensor);
        }
        // Stack all blocks along dim=1 → [2, num_blocks, H, B, D]
        auto layer_tensor = torch::stack(blocks_for_layer, 1).contiguous();
        src_tensors.push_back(layer_tensor);
    }


    // 8. Build mapping tensor (src index → dst block_id)
    std::vector<int64_t> mapping_vec;
    mapping_vec.reserve(num_read_blocks * 2);
    for (int i = 0; i < num_read_blocks; i++) {
        mapping_vec.push_back(i);                   // src index in stacked tensor
        mapping_vec.push_back(block_ids_list[i]);   // target block_id
    }
    auto src_to_dst = torch::from_blob(
        mapping_vec.data(),
        {(int64_t)num_read_blocks, 2},
        torch::TensorOptions().dtype(torch::kInt64).device(torch::kCPU)
    ).clone();

    // 9. Call backend directly
    swap_blocks_multi_layer(src_tensors, dst_tensors, src_to_dst);

    return true;
}

// Async Storage → GPU transfer (GET)
bool transfer_async_get_ext(
    int job_id,
    std::vector<std::string> source_files,
    std::vector<std::vector<int64_t>> all_block_ids,
    std::vector<torch::Tensor> dst_tensors,
    int gpu_blocks_per_file)
{
    init_performance_resources();

    std::lock_guard<std::mutex> lock(jobs_mutex);
    auto job_state = std::make_unique<JobState>();
    job_state->total_tasks = source_files.size();

    for (size_t i = 0; i < source_files.size(); i++) {
        std::string src_file = source_files[i];
        auto block_ids = all_block_ids[i];
        auto future = g_io_pool->enqueue([=, job_state = job_state.get()]() -> bool {

            // Get dedicated stream for this thread
            if (!thread_stream.has_value()) {
                thread_stream = g_streams[thread_stream_idx % g_streams.size()];
            }

            auto current_stream = at::cuda::getCurrentCUDAStream();
            at::cuda::setCurrentCUDAStream(*thread_stream);

            bool stage1_ok = false;
            torch::Tensor host_buf;

            // -------------------------
            // Stage 1: File → pinned CPU
            // -------------------------
            try {
                host_buf = read_file_to_pinned_tensor(src_file);
                stage1_ok = true;
            } catch (const std::exception& e) {
                std::cerr << "[ERROR] Stage1 read_file_to_pinned_tensor failed for "
                        << src_file << ": " << e.what() << std::endl;
            } catch (...) {
                std::cerr << "[ERROR] Stage1 unknown failure for " << src_file << std::endl;
            }

            // -------------------------
            // Stage 2: CPU → GPU copy
            // -------------------------
            bool ok = false;
            if (stage1_ok) {
                try {
                    ok = copy_and_swap_gpu_tensors_ext(
                        host_buf, block_ids, dst_tensors, gpu_blocks_per_file);

                    hipError_t err = hipStreamSynchronize(thread_stream->stream());
                    if (err != hipSuccess) {
                        std::cerr << "[ERROR] hipStreamSynchronize failed: "
                                << hipGetErrorString(err) << std::endl;
                        ok = false;
                    }

                } catch (const std::exception& e) {
                    std::cerr << "[ERROR] Stage2 copy_and_swap failed for " << src_file
                            << ": " << e.what() << std::endl;
                } catch (...) {
                    std::cerr << "[ERROR] Stage2 unknown failure for " << src_file << std::endl;
                }
            }

            // -------------------------
            // Final cleanup & accounting
            // -------------------------
            at::cuda::setCurrentCUDAStream(current_stream);
            job_state->completed_tasks.fetch_add(1);
            if (!ok) job_state->all_success = false;
            return ok;
        });

        job_state->futures.push_back(future.share());
    }

    jobs[job_id] = std::move(job_state);
    return true;
}

// -------------------------------
// PYBIND11 module
// -------------------------------
PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("init_performance_resources", &init_performance_resources,
          py::arg("io_threads") = 0,
          py::arg("pinned_buffer_size_mb") = 256,
          py::arg("max_pinned_memory_gb") = 50);
    m.def("cleanup_performance_resources", &cleanup_performance_resources);
    m.def("get_finished_ext", &get_finished_ext);

    m.def("transfer_async_put_ext", &transfer_async_put_ext,
          "Async transfer  GPU-> Storage",
          py::arg("job_id"),
          py::arg("target_files"),
          py::arg("all_src_tensors"),
          py::arg("all_block_ids"));

    m.def("transfer_async_get_ext", &transfer_async_get_ext,
      "Async GET transfer from Storage → GPU",
      py::arg("job_id"),
      py::arg("source_files"),
      py::arg("all_block_ids"),
      py::arg("dst_tensors"),
      py::arg("gpu_blocks_per_file"));
    m.def("swap_blocks_multi_layer", &swap_blocks_multi_layer);
}